#include "hip/hip_runtime.h"
#pragma once

#include <cmath>
#include <iostream>
#include <memory>
#include <vector>

#include "constant.h"
#include "tensor.h"

template <typename T>
ushionn::Tensor::Tensor(std::initializer_list<size_t> shapes, const std::vector<T>& data)
    : dtype_(TypeToEnum<T>::value),
      shape_(shape),
      data_(new T[data.size()]),
      data_size_(data.size()),
      shape_size_(shapes.size())
{
    std::copy(data.begin(), data.end(), static_cast<T*>(data_.get()));
}

void ushionn::Tensor::CUDA()
{
    if (device_ == Device::CUDA) return;
    void* ptr = nullptr;

    hipMalloc(&ptr, data_size_ * GetDTypeSize());
    auto err_code = hipMemcpy(ptr, data_.get(), data_size_ * GetDTypeSize(), hipMemcpyHostToDevice);
    device_ = Device::CUDA;

    if (err_code != hipSuccess)
    {
        std::cerr << "Error : failed to copy Tensor from host to device, Error Code : " << err_code << std::endl;
        hipFree(ptr);
    }
    else
    {
        data_.reset(ptr);
    }
}

void ushionn::Tensor::CPU()
{
    if (device_ == Device::CPU) return;
    void* ptr = nullptr;

    AllocCPUArray(ptr, data_size_);
    auto err_code = hipMemcpy(ptr, data_.get(), data_size_ * GetDTypeSize(), hipMemcpyDeviceToHost);
    device_ = Device::CPU;

    if (err_code != hipSuccess)
    {
        std::cerr << "Error : failed to copy Tensor from device to host, Error Code : " << err_code << std::endl;
    }
    else
    {
        hipFree(data_.get());
        data_.release();
        data_.reset(ptr);
    }
}

ushionn::Device ushionn::Tensor::GetDevice() const
{
    return device_;
}

template <typename T>
T ushionn::Tensor::Index(std::initializer_list<size_t> indexList)
{
    std::vector<size_t> tmp(indexList);
    if (tmp.size() != shapeSize_)
    {
        std::cerr << "Error : Given index list do not match dimension size" << std::endl;
        throw "given index list do not match dimension size";
    }

    int idx = 0;

    for (int i = 0; i < shapeSize_; ++i)
    {
        int multiple = 1;
        for (int j = shapeSize_ - 1; j > i; --j) multiple *= shape_[j];
        idx += tmp[i] * multiple;
    }

    return data_.get()[idx];
}

bool ushionn::Tensor::SetDims(std::initializer_list<size_t> dimList)
{
    std::vector<size_t> tmp(dimList);
    int size = 1;
    for (const auto& dim : tmp) size *= dim;
    if (size != shape_size_) return false;
    shape_.assign(dimList);
    shape_size_ = tmp.size();
    return true;
}

size_t ushionn::Tensor::GetDTypeSize()
{
    if (dtype_ == DataType::FLOAT32)
        return sizeof(float);
    else if (dtype_ == DataType::FLOAT64)
        return sizeof(double);
    else if (dtype_ == DataType::INT32)
        return sizeof(int);
}

void ushionn::Tensor::AllocCPUArray(void* ptr, size_t size)
{
    if (dtype_ == DataType::FLOAT32)
        ptr = new float[size];
    else if (dtype_ == DataType::FLOAT64)
        ptr = new double[size];
    else if (dtype_ == DataType::INT32)
        ptr = new int[size];
}

#ifdef USE_CUDNN
#else  // #TODO 나중에 수정
template <typename T, typename S>
__global__ void MultiplyCUDA1D(const T* src, const S target, T* out, const size_t dimX)
{
    const int tid = Grid1DTID(blockIdx.x, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX) return;
    out[tid] = src[tid] * target;
}

template <typename T, typename S>
__global__ void MultiplyCUDA2D(const T* src, const S target, T* out, const size_t dimX, const size_t dimY)
{
    const int tid = Grid2DTID(blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z);
    const int tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const int tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX || tDimY >= dimY) return;
    out[tid] = src[tid] * target;
}

template <typename T, typename S>
__global__ void MultiplyCUDA3D(const T* src, const S target, T* out, const size_t dimX, const size_t dimY,
                               const size_t dimZ)
{
    const int tid = Grid3DTID(blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimZ = GDim_Z * blockIdx.z + threadIdx.z;
    const size_t tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;

    if (tDimX >= dimX || tDimY >= dimY || tDimZ >= dimZ) return;
    out[tid] = src[tid] * target;
}

template <typename T>
__global__ void AddCUDA1D(const T* src, const T* target, T* out, const size_t dimX)
{
    const int tid = Grid1DTID(blockIdx.x, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX) return;
    out[tid] = src[tid] + target[tid];
}

template <typename T>
__global__ void AddCUDA2D(const T* src, const T* target, T* out, const size_t dimX, const size_t dimY)
{
    const int tid = Grid2DTID(blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z);
    const int tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const int tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX || tDimY >= dimY) return;
    out[tid] = src[tid] + target[tid];
}

template <typename T>
__global__ void AddCUDA3D(const T* src, const T* target, T* out, const size_t dimX, const size_t dimY,
                          const size_t dimZ)
{
    const int tid = Grid3DTID(blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimZ = GDim_Z * blockIdx.z + threadIdx.z;
    const size_t tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;

    if (tDimX >= dimX || tDimY >= dimY || tDimZ >= dimZ) return;
    out[tid] = src[tid] + target[tid];
}

template <typename T>
template <typename S>
void ushionn::Tensor<T>::Multiply(const S x)
{
    if (device_ == Device::CUDA)
    {
        if (dimSize_ == 1 && dataSize_ <= 1024)
        {
            MultiplyCUDA1D<T, S><<<dim3(1, 1, 1), dim3(dims_[0], 1, 1)> > >(data_.get(), x, data_.get(), dims_[0]);
        }
        else if (dimSize_ == 1 && dataSize_ > 1024)
        {
            MultiplyCUDA1D<T, S><<<dim3(ceil(dataSize_ / 1024.f), 1, 1), dim3(blockSize1D, 1, 1)> > >(
                data_.get(), x, data_.get(), dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ <= 1024)
        {
            MultiplyCUDA2D<T, S>
                <<<dim3(1, 1, 1), dim3(dims_[1], dims_[0], 1)> > >(data_.get(), x, data_.get(), dims_[1], dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ > 1024)
        {
            MultiplyCUDA2D<T, S>
                <<<dim3(ceil(dims_[1] / static_cast<float>(kblockSize2D)),
                        ceil(dims_[0] / static_cast<float>(kblockSize2D))),
                   dim3(blockSize2D, blockSize2D, 1)> > >(data_.get(), x, data_.get(), dims_[1], dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ <= 1024)
        {
            MultiplyCUDA3D<T, S><<<dim3(1, 1, 1), dim3(dims_[2], dims_[1], dims_[0])> > >(data_.get(), x, data_.get(),
                                                                                          dims_[2], dims_[1], dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ > 1024)
        {
            MultiplyCUDA3D<T, S><<<dim3(ceil(dims_[2] / static_cast<float>(kblockSize3DX)),
                                        ceil(dims_[1] / static_cast<float>(kblockSize3DYZ)),
                                        ceil(dims_[0] / static_cast<float>(kblockSize3DYZ))),
                                   dim3(blockSize3DX, blockSize3DYZ, blockSize3DYZ)> > >(data_.get(), x, data_.get(),
                                                                                         dims_[2], dims_[1], dims_[0]);
        }
        else
        {
            std::cerr << "Error : Tensor dim length must be less than 4" << std::endl;
            throw "Tensor dim length must be less than 4";
        }
    }
    else if (device_ == Device::CPU)
    {
        for (int i = 0; i < dataSize_; ++i) data_.get()[i] *= x;
    }
}

template <typename T>
void ushionn::Tensor<T>::Add(const Tensor& x)
{
    if (device_ == Device::CUDA && x.getDevice() == Device::CUDA && dims_ == x.dims_)
    {
        if (dimSize_ == 1 && dataSize_ <= 1024)
        {
            AddCUDA1D<T><<<dim3(1, 1, 1), dim3(dims_[0], 1, 1)> > >(data_.get(), x.data_.get(), data_.get(), dims_[0]);
        }
        else if (dimSize_ == 1 && dataSize_ > 1024)
        {
            AddCUDA1D<T><<<dim3(ceil(dataSize_ / 1024.f), 1, 1), dim3(blockSize1D, 1, 1)> > >(
                data_.get(), x.data_.get(), data_.get(), dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ <= 1024)
        {
            AddCUDA2D<T><<<dim3(1, 1, 1), dim3(dims_[1], dims_[0], 1)> > >(data_.get(), x.data_.get(), data_.get(),
                                                                           dims_[1], dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ > 1024)
        {
            AddCUDA2D<T><<<dim3(ceil(dims_[1] / static_cast<float>(kblockSize2D)),
                                ceil(dims_[0] / static_cast<float>(kblockSize2D))),
                           dim3(kblockSize2D, kblockSize2D, 1)> > >(data_.get(), x.data_.get(), data_.get(), dims_[1],
                                                                    dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ <= 1024)
        {
            AddCUDA3D<T><<<dim3(1, 1, 1), dim3(dims_[2], dims_[1], dims_[0])> > >(
                data_.get(), x.data_.get(), data_.get(), dims_[2], dims_[1], dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ > 1024)
        {
            AddCUDA3D<T><<<dim3(ceil(dims_[2] / static_cast<float>(kblockSize3DX)),
                                ceil(dims_[1] / static_cast<float>(kblockSize3DYZ)),
                                ceil(dims_[0] / static_cast<float>(kblockSize3DYZ))),
                           dim3(kblockSize3DX, kblockSize3DYZ, kblockSize3DYZ)> > >(
                data_.get(), x.data_.get(), data_.get(), dims_[2], dims_[1], dims_[0]);
        }
        else
        {
            std::cerr << "Error : Tensor dim must be 1, 2, 3" << std::endl;
            throw "Tensor dim must be 1, 2, 3";
        }
    }
    else if (device_ == Device::CPU && x.getDevice() == Device::CPU && dims_ == x.dims_)
    {
        for (int i = 0; i < dataSize_; ++i) data_.get()[i] += x.data_.get()[i];
    }
    else if (device_ == x.device_ && dims_ != x.dims_)
    {
        std::cerr << "Error : Tensors need to be in the same dimension" << std::endl;
        throw "Tensors need to be in the same dimension";
    }
    else
    {
        std::cerr << "Error : Tensors need to be in the same device" << std::endl;
        throw "Tensors need to be in the same device";
    }
}

template void ushionn::Tensor<int>::Multiply<int>(int);
template void ushionn::Tensor<int>::Multiply<float>(float);
template void ushionn::Tensor<float>::Multiply<int>(int);
template void ushionn::Tensor<float>::Multiply<float>(float);

#endif
