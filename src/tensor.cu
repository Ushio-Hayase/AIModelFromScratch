#include "hip/hip_runtime.h"
#pragma once

#include <cmath>
#include <iostream>
#include <memory>
#include <vector>

#include "constant.cuh"
#include "tensor.cuh"

template <typename T>
ushionn::Tensor::Tensor(std::initializer_list<size_t> shapes, const std::vector<T>& data)
    : dtype_(TypeToEnum<T>::value),
      shape_(shape),
      data_(new T[data.size()]),
      dataSize_(data.size()),
      shapeSize_(shapes.size())
{
    std::copy(data.begin(), data.end(), static_cast<T*>(data_.get()));
}

void ushionn::Tensor::CUDA()
{
    if (device_ == Device::CUDA) return;
    void* ptr = nullptr;

    hipMalloc(&ptr, dataSize_ * GetDTypeSize());
    auto errCode = hipMemcpy(ptr, data_.get(), dataSize_ * GetDTypeSize(), hipMemcpyHostToDevice);
    device_ = Device::CUDA;

    if (errCode != hipSuccess)
    {
        std::cerr << "Error : failed to copy Tensor from host to device, Error Code : " << errCode << std::endl;
        hipFree(ptr);
    }
    else
    {
        data_.reset(ptr);
    }
}

void ushionn::Tensor::CPU()
{
    if (device_ == Device::CPU) return;
    void* ptr = nullptr;

    AllocCPUArray(ptr, dataSize_);
    auto errCode = hipMemcpy(ptr, data_.get(), dataSize_ * GetDTypeSize(), hipMemcpyDeviceToHost);
    device_ = Device::CPU;

    if (errCode != hipSuccess)
    {
        std::cerr << "Error : failed to copy Tensor from device to host, Error Code : " << errCode << std::endl;
    }
    else
    {
        hipFree(data_.get());
        data_.release();
        data_.reset(ptr);
    }
}

ushionn::Device ushionn::Tensor::GetDevice() const
{
    return device_;
}

template <typename T>
T ushionn::Tensor::Index(std::initializer_list<size_t> indexList)
{
    std::vector<size_t> tmp(indexList);
    if (tmp.size() != shapeSize_)
    {
        std::cerr << "Error : Given index list do not match dimension size" << std::endl;
        throw "given index list do not match dimension size";
    }

    int idx = 0;

    for (int i = 0; i < shapeSize_; ++i)
    {
        int multiple = 1;
        for (int j = shapeSize_ - 1; j > i; --j) multiple *= shape_[j];
        idx += tmp[i] * multiple;
    }

    return data_.get()[idx];
}

bool ushionn::Tensor::SetDims(std::initializer_list<size_t> dimList)
{
    std::vector<size_t> tmp(dimList);
    int size = 1;
    for (const auto& dim : tmp) size *= dim;
    if (size != shapeSize_) return false;
    shape_.assign(dimList);
    shapeSize_ = tmp.size();
    return true;
}

size_t ushionn::Tensor::GetDTypeSize()
{
    if (dtype_ == DataType::FLOAT32)
        return sizeof(float);
    else if (dtype_ == DataType::FLOAT64)
        return sizeof(double);
    else if (dtype_ == DataType::INT32)
        return sizeof(int);
}

void ushionn::Tensor::AllocCPUArray(void* ptr, size_t size)
{
    if (dtype_ == DataType::FLOAT32)
        ptr = new float[size];
    else if (dtype_ == DataType::FLOAT64)
        ptr = new double[size];
    else if (dtype_ == DataType::INT32)
        ptr = new int[size];
}

#ifdef USE_CUDNN
#else  // #TODO 나중에 수정
template <typename T, typename S>
__global__ void MultiplyCUDA1D(const T* src, const S target, T* out, const size_t dimX)
{
    const int tid = Grid1DTID(blockIdx.x, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX) return;
    out[tid] = src[tid] * target;
}

template <typename T, typename S>
__global__ void MultiplyCUDA2D(const T* src, const S target, T* out, const size_t dimX, const size_t dimY)
{
    const int tid = Grid2DTID(blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z);
    const int tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const int tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX || tDimY >= dimY) return;
    out[tid] = src[tid] * target;
}

template <typename T, typename S>
__global__ void MultiplyCUDA3D(const T* src, const S target, T* out, const size_t dimX, const size_t dimY,
                               const size_t dimZ)
{
    const int tid = Grid3DTID(blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimZ = GDim_Z * blockIdx.z + threadIdx.z;
    const size_t tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;

    if (tDimX >= dimX || tDimY >= dimY || tDimZ >= dimZ) return;
    out[tid] = src[tid] * target;
}

template <typename T>
__global__ void AddCUDA1D(const T* src, const T* target, T* out, const size_t dimX)
{
    const int tid = Grid1DTID(blockIdx.x, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX) return;
    out[tid] = src[tid] + target[tid];
}

template <typename T>
__global__ void AddCUDA2D(const T* src, const T* target, T* out, const size_t dimX, const size_t dimY)
{
    const int tid = Grid2DTID(blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z);
    const int tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const int tDimX = GDim_X * blockIdx.x + threadIdx.x;
    if (tDimX >= dimX || tDimY >= dimY) return;
    out[tid] = src[tid] + target[tid];
}

template <typename T>
__global__ void AddCUDA3D(const T* src, const T* target, T* out, const size_t dimX, const size_t dimY,
                          const size_t dimZ)
{
    const int tid = Grid3DTID(blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
    const size_t tDimZ = GDim_Z * blockIdx.z + threadIdx.z;
    const size_t tDimY = GDim_Y * blockIdx.y + threadIdx.y;
    const size_t tDimX = GDim_X * blockIdx.x + threadIdx.x;

    if (tDimX >= dimX || tDimY >= dimY || tDimZ >= dimZ) return;
    out[tid] = src[tid] + target[tid];
}

template <typename T>
template <typename S>
void ushionn::Tensor<T>::Multiply(const S x)
{
    if (device_ == Device::CUDA)
    {
        if (dimSize_ == 1 && dataSize_ <= 1024)
        {
            MultiplyCUDA1D<T, S><<<dim3(1, 1, 1), dim3(dims_[0], 1, 1)> > >(data_.get(), x, data_.get(), dims_[0]);
        }
        else if (dimSize_ == 1 && dataSize_ > 1024)
        {
            MultiplyCUDA1D<T, S><<<dim3(ceil(dataSize_ / 1024.f), 1, 1), dim3(blockSize1D, 1, 1)> > >(
                data_.get(), x, data_.get(), dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ <= 1024)
        {
            MultiplyCUDA2D<T, S>
                <<<dim3(1, 1, 1), dim3(dims_[1], dims_[0], 1)> > >(data_.get(), x, data_.get(), dims_[1], dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ > 1024)
        {
            MultiplyCUDA2D<T, S>
                <<<dim3(ceil(dims_[1] / static_cast<float>(blockSize2D)),
                        ceil(dims_[0] / static_cast<float>(blockSize2D))),
                   dim3(blockSize2D, blockSize2D, 1)> > >(data_.get(), x, data_.get(), dims_[1], dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ <= 1024)
        {
            MultiplyCUDA3D<T, S><<<dim3(1, 1, 1), dim3(dims_[2], dims_[1], dims_[0])> > >(data_.get(), x, data_.get(),
                                                                                          dims_[2], dims_[1], dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ > 1024)
        {
            MultiplyCUDA3D<T, S><<<dim3(ceil(dims_[2] / static_cast<float>(blockSize3DX)),
                                        ceil(dims_[1] / static_cast<float>(blockSize3DYZ)),
                                        ceil(dims_[0] / static_cast<float>(blockSize3DYZ))),
                                   dim3(blockSize3DX, blockSize3DYZ, blockSize3DYZ)> > >(data_.get(), x, data_.get(),
                                                                                         dims_[2], dims_[1], dims_[0]);
        }
        else
        {
            std::cerr << "Error : Tensor dim length must be less than 4" << std::endl;
            throw "Tensor dim length must be less than 4";
        }
    }
    else if (device_ == Device::CPU)
    {
        for (int i = 0; i < dataSize_; ++i) data_.get()[i] *= x;
    }
}

template <typename T>
void ushionn::Tensor<T>::Add(const Tensor& x)
{
    if (device_ == Device::CUDA && x.getDevice() == Device::CUDA && dims_ == x.dims_)
    {
        if (dimSize_ == 1 && dataSize_ <= 1024)
        {
            AddCUDA1D<T><<<dim3(1, 1, 1), dim3(dims_[0], 1, 1)> > >(data_.get(), x.data_.get(), data_.get(), dims_[0]);
        }
        else if (dimSize_ == 1 && dataSize_ > 1024)
        {
            AddCUDA1D<T><<<dim3(ceil(dataSize_ / 1024.f), 1, 1), dim3(blockSize1D, 1, 1)> > >(
                data_.get(), x.data_.get(), data_.get(), dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ <= 1024)
        {
            AddCUDA2D<T><<<dim3(1, 1, 1), dim3(dims_[1], dims_[0], 1)> > >(data_.get(), x.data_.get(), data_.get(),
                                                                           dims_[1], dims_[0]);
        }
        else if (dimSize_ == 2 && dataSize_ > 1024)
        {
            AddCUDA2D<T>
                <<<dim3(ceil(dims_[1] / static_cast<float>(blockSize2D)),
                        ceil(dims_[0] / static_cast<float>(blockSize2D))),
                   dim3(blockSize2D, blockSize2D, 1)> > >(data_.get(), x.data_.get(), data_.get(), dims_[1], dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ <= 1024)
        {
            AddCUDA3D<T><<<dim3(1, 1, 1), dim3(dims_[2], dims_[1], dims_[0])> > >(
                data_.get(), x.data_.get(), data_.get(), dims_[2], dims_[1], dims_[0]);
        }
        else if (dimSize_ == 3 && dataSize_ > 1024)
        {
            AddCUDA3D<T><<<dim3(ceil(dims_[2] / static_cast<float>(blockSize3DX)),
                                ceil(dims_[1] / static_cast<float>(blockSize3DYZ)),
                                ceil(dims_[0] / static_cast<float>(blockSize3DYZ))),
                           dim3(blockSize3DX, blockSize3DYZ, blockSize3DYZ)> > >(
                data_.get(), x.data_.get(), data_.get(), dims_[2], dims_[1], dims_[0]);
        }
        else
        {
            std::cerr << "Error : Tensor dim must be 1, 2, 3" << std::endl;
            throw "Tensor dim must be 1, 2, 3";
        }
    }
    else if (device_ == Device::CPU && x.getDevice() == Device::CPU && dims_ == x.dims_)
    {
        for (int i = 0; i < dataSize_; ++i) data_.get()[i] += x.data_.get()[i];
    }
    else if (device_ == x.device_ && dims_ != x.dims_)
    {
        std::cerr << "Error : Tensors need to be in the same dimension" << std::endl;
        throw "Tensors need to be in the same dimension";
    }
    else
    {
        std::cerr << "Error : Tensors need to be in the same device" << std::endl;
        throw "Tensors need to be in the same device";
    }
}

template void ushionn::Tensor<int>::Multiply<int>(int);
template void ushionn::Tensor<int>::Multiply<float>(float);
template void ushionn::Tensor<float>::Multiply<int>(int);
template void ushionn::Tensor<float>::Multiply<float>(float);

#endif
